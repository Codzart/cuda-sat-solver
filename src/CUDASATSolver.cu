#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include "CUDASATSolver.h"
#include "config.h"

extern void remove_literal_from_clause(uint32_t index_to_remove, clause &c);
extern void remove_clause_from_formula(uint32_t index_to_remove, formula &f);
extern literal choose_next_literal(formula& f, assignment& partial);

assignment* check_sat_given_partial_assignment(formula& f, assignment& partial, int level); // DONE
void process_unit_clauses(formula& f, assignment& partial); // DONE
assignment* assign_pure_literals(formula& f, assignment& partial); // DONE
void unit_propagate(literal l, formula& f); // DONE
assignment* check_sat_status(formula& f, assignment& partial); // DONE

static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__host__ __device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */

void unit_propagate(literal l, formula& f)
{
	for(int i=0 ; i<f.nbOfClauses ; i++)
	{
		clause& c = f.clauses[i];
		for(int j=0 ; j<c.length ; j++)
		{
			if(c.literals[j]==l)
			{
				remove_clause_from_formula(i,f);
				i--;
				break;
			}
			if(c.literals[j]==-l)
			{
				remove_literal_from_clause(j,c);
				j--;
			}
		}
	}
}

void process_unit_clauses(formula& f, assignment& partial)
{
	for(int i=0 ; i<f.nbOfClauses ; i++)
	{
		clause& c = f.clauses[i];
		if(c.length == 1)
		{
			literal& l = c.literals[0];

            if(partial.literals[abs(l)]==0)
            {
                if(l>0)
                {
                    partial.literals[l] = 1;
                }
                else
                {
                    partial.literals[-l] = -1;
                }
                unit_propagate(l,f);

                process_unit_clauses(f,partial);
            }
		}
	}
}

assignment* assign_pure_literals(formula& f, assignment& partial)
{
	bool* positive = new bool[f.nbOfVariables+1];
	bool* negative = new bool[f.nbOfVariables+1];

	for(int i=0 ; i<f.nbOfVariables+1 ; i++)
	{
		positive[i] = negative[i] = false;
	}

	for(int i=0 ; i<f.nbOfClauses ; i++)
	{
		clause& c = f.clauses[i];
		for(int j=0 ; j<c.length ; j++)
		{
			literal& l = c.literals[j];
			if(l>0)
			{
				positive[l] = true;
			}
			else
			{
				negative[-l] = true;
			}
		}
	}

	bool changed = false;

	for(int i=1 ; i<f.nbOfVariables+1 ; i++)
	{
		if(partial.literals[i] == 0 && positive[i] ^ negative[i])
		{
			if(positive[i])
			{
				partial.literals[i] = 1;
				unit_propagate(i,f);
				changed = true;
			}
			else
			{
				partial.literals[i] = -1;
				unit_propagate(-i,f);
				changed = true;
			}

		}
	}

    delete[] positive;
    delete[] negative;

	if(changed)
	{
        assignment* res = check_sat_status(f,partial);
        if(res!=NULL)
        {
        	return res;
        }

		process_unit_clauses(f,partial);
		assign_pure_literals(f,partial);
	}

	return NULL;
}

assignment* check_sat_status(formula& f, assignment& partial)
{
    if(f.nbOfClauses==0)
	{
    	assignment* res = new assignment;
    	*res = deepcopy(partial);
    	res->unsat = false;
    	return res;
	}

	for(int i=0 ; i<f.nbOfClauses ; i++)
	{
		clause& c = f.clauses[i];
		if(c.length == 0)
		{
			assignment* res = new assignment;
			res->unsat = true;
			return res;
		}
	}
	return NULL;
}

#ifdef PRINT_BACKTRACKING
static int min_backtrack_level = -1;
#endif

assignment* check_sat_given_partial_assignment(formula& f, assignment& partial, int level)
{
	assignment* res;

	res = check_sat_status(f,partial);
	if(res!=NULL)
	{
		return res;
	}

	process_unit_clauses(f,partial);

	res = assign_pure_literals(f,partial);
	if(res!=NULL)
	{
		return res;
	}

	res = check_sat_status(f,partial);
	if(res!=NULL)
	{
		return res;
	}

    formula fbis;

    literal l = choose_next_literal(f, partial);
    if(l==0)
    {
    	assignment* res = new assignment;
		res->unsat = true;
		return res;
    }

    if(l>0)
    {
    	assignment a = deepcopy(partial);
		a.literals[l] = 1;
		fbis = copy(f);
		unit_propagate(l,fbis);
		res = check_sat_given_partial_assignment(fbis,a,level+1);
		if(!res->unsat)
		{
			return res;
		}

		// Left subtree was UNSAT

#ifdef PRINT_BACKTRACKING
		if(min_backtrack_level == -1 || min_backtrack_level>level)
		{
			min_backtrack_level = level;
			printf("Backtracking to level %d\n",level);
		}
#endif
		partial.literals[l] = -1;
		dealloc(fbis);
		unit_propagate(-l,f);
		return check_sat_given_partial_assignment(f,partial,level+1);
    }
    else
    {
    	assignment a = deepcopy(partial);
		a.literals[l] = -1;
		fbis = copy(f);
		unit_propagate(l,fbis);
		res = check_sat_given_partial_assignment(fbis,a,level+1);

		if(!res->unsat)
		{
			return res;
		}

		// Left subtree was UNSAT

#ifdef PRINT_BACKTRACKING
		if(min_backtrack_level == -1 || min_backtrack_level>level)
		{
			min_backtrack_level = level;
			printf("Backtracking to level %d\n",level);
		}
#endif
		partial.literals[l] = 1;
		dealloc(fbis);
		unit_propagate(-l,f);
		return check_sat_given_partial_assignment(f,partial,level+1);
    }
}

assignment* cuda_check_sat(formula& f) {
	assignment partial;
	partial.length = f.nbOfVariables;
	partial.literals = new literal[partial.length];
	for(int i=0 ; i<partial.length ; i++)
	{
		partial.literals[i] = 0;
	}
	return check_sat_given_partial_assignment(f,partial,0);
}
